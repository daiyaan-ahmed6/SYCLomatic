
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *h, __half2 *h2, float *b,
                     float2 *b2) {
  // Start
  /* 1 */ __ldg(h /*__half **/);
  /* 2 */ __ldg(h2 /*__half2 **/);
  /* 3 */ __ldg(b /*__float **/);
  /* 4 */ __ldg(b2 /*__float2 **/);
  // End
}
