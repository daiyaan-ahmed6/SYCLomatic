
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const double *alpha, const double *a, int lda, const double *x,
          int incx, const double *beta, double *y, int incy) {
  // Start
  hipblasDsymv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const double **/, a /*const double **/,
              lda /*int*/, x /*const double **/, incx /*int*/,
              beta /*const double **/, y /*double **/, incy /*int*/);
  // End
}
