
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *x, int incx, float *res) {
  // Start
  hipblasSasum(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
              incx /*int*/, res /*float **/);
  // End
}
