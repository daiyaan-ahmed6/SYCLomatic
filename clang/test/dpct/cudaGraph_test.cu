// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --format-range=none --use-experimental-features=graph -out-root %T/cudaGraph_test %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cudaGraph_test/cudaGraph_test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST %T/cudaGraph_test/cudaGraph_test.dp.cpp -o %T/cudaGraph_test/cudaGraph_test.dp.o %}

#ifndef BUILD_TEST
#include <hip/hip_runtime.h>

int main() {
  // CHECK: dpct::experimental::command_graph_t graph;
  // CHECK-NEXT: dpct::experimental::command_graph_t *graph2;
  // CHECK-NEXT: dpct::experimental::command_graph_t **graph3;
  hipGraph_t graph;
  hipGraph_t *graph2;
  hipGraph_t **graph3;

  // CHECK: dpct::experimental::command_graph_t graph4[10];
  hipGraph_t graph4[10];

  // CHECK: dpct::experimental::command_graph_t graph5, *graph6, **graph7;
  hipGraph_t graph5, *graph6, **graph7;

  // CHECK: dpct::experimental::command_graph_exec_t execGraph;
  // CHECK-NEXT: dpct::experimental::command_graph_exec_t *execGraph2;
  // CHECK-NEXT: dpct::experimental::command_graph_exec_t **execGraph3;
  hipGraphExec_t execGraph;
  hipGraphExec_t *execGraph2;
  hipGraphExec_t **execGraph3;

  // CHECK: dpct::experimental::command_graph_exec_t execGraph4[10];
  hipGraphExec_t execGraph4[10];

  // CHECK: dpct::experimental::command_graph_exec_t execGraph5, *execGraph6, **execGraph7;
  hipGraphExec_t execGraph5, *execGraph6, **execGraph7;

  return 0;
}

#endif // BUILD_TEST
